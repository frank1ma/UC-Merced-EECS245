#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <sys/time.h>

 
#define BLOCKSIZE 16
 
typedef struct{
  size_t width;
  size_t height;
  size_t stride;
  float * elements;
 
}matrix_t;
 
__device__ float getElement(const matrix_t * mat, int row, int col){
  return mat->elements[mat->stride * row + col];
}
 
__device__ void setElement(matrix_t * mat, int row, int col, float value){
  mat->elements[mat->stride * row + col] = value;
}
 
__device__ matrix_t getSubMatrix(matrix_t mat, int row, int col){
  matrix_t matAns;
  matAns.width = BLOCKSIZE;
  matAns.height = BLOCKSIZE;
  matAns.stride = mat.stride;
     
        matAns.elements = mat.elements +  row * BLOCKSIZE * mat.stride + col * BLOCKSIZE;
 
	return matAns;
}

__global__ void matMulKernel(matrix_t ma, matrix_t mb, matrix_t mc){
  float cValue = 0;   
  
  int blockRow = blockIdx.y;
  int blockCol = blockIdx.x;
 
  int row = threadIdx.y;
  int col = threadIdx.x;
  for (int subIdx = 0; subIdx < ma.width / BLOCKSIZE; ++subIdx){
      __shared__ float s_subMa[BLOCKSIZE][BLOCKSIZE];
      __shared__ float s_subMb[BLOCKSIZE][BLOCKSIZE];
 
      matrix_t subMatA = getSubMatrix(ma, blockRow, subIdx);
      matrix_t subMatB = getSubMatrix(mb, subIdx, blockCol);
 
      s_subMa[row][col] = getElement(&subMatA, row, col);
      s_subMb[row][col] = getElement(&subMatB, row, col);
 
      __syncthreads();
 
      for (int k = 0; k < BLOCKSIZE; ++k) {
	       cValue += (s_subMa[row][k] * s_subMb[k][col]);
      }
 
      __syncthreads();
    }
 
  matrix_t subMatC = getSubMatrix(mc, blockRow, blockCol);
  setElement(&subMatC, row, col, cValue);
}

void callMatMulKernel(){
  matrix_t matA;
  matA.width = 1024;
  matA.height = 2048;
  matA.stride = matA.width;
  matA.elements = (float *)malloc(matA.width * matA.height * sizeof(float));
 
  matrix_t matB;
  matB.width = 1024;
  matB.height = 1024;
  matB.stride = matB.width;
  matB.elements = (float *)malloc(matB.width * matB.height * sizeof(float));
 
  matrix_t matAns;
  matAns.width = matB.width;
  matAns.height = matA.height;
  matAns.stride = matAns.width;
  matAns.elements = (float *)malloc(matAns.width * matAns.height * sizeof(float));
  memset(matAns.elements,0, matAns.width * matAns.height * sizeof(float));
 
  for (int i = 0; i < matA.width * matA.height; ++i){
    matA.elements[i] = i * 0.1;
  }
  for (int i = 0; i < matB.width * matB.height; ++i){
    matB.elements[i] = i * 0.1;
  }
 
  matrix_t d_matA;
  d_matA.width = matA.width;
  d_matA.height = matA.height;
  d_matA.stride = matA.stride;
    size_t size = d_matA.width *  d_matA.height * sizeof(float);
    hipMalloc(&d_matA.elements, size);
    hipMemcpy(d_matA.elements, matA.elements, size, hipMemcpyHostToDevice);
 
    matrix_t d_matB;
    d_matB.width = matB.width;
    d_matB.height = matB.height;
    d_matB.stride = matB.stride;
    size = d_matB.width * d_matB.height * sizeof(float);
    hipMalloc(&d_matB.elements, size);
    hipMemcpy(d_matB.elements, matB.elements, size, hipMemcpyHostToDevice);
 
    matrix_t d_matC;
    d_matC.width = matAns.width;
    d_matC.height = matAns.height;
    d_matC.stride = matAns.stride;
    hipMalloc(&d_matC.elements,d_matC.width * d_matC.height * sizeof(float));
 
    dim3 threadsPerBlock(BLOCKSIZE, BLOCKSIZE);
    dim3 blocksPerGrid(matB.width / threadsPerBlock.x, matA.height / threadsPerBlock.y);
    matMulKernel<<<blocksPerGrid, threadsPerBlock>>>(d_matA, d_matB, d_matC);
 
    hipMemcpy(matAns.elements, d_matC.elements,d_matC.width * d_matC.height * sizeof(float),hipMemcpyDeviceToHost);
 
    hipFree(d_matA.elements);
    hipFree(d_matB.elements);
    hipFree(d_matC.elements);
 
    free(matA.elements);
    free(matB.elements);
    free(matAns.elements);
    return;
}
 
int main(){
  struct timeval tv1, tv0;
  gettimeofday(&tv0, NULL);
  callMatMulKernel();
  gettimeofday(&tv1, NULL);
  printf("time: %lf\n", double(tv1.tv_usec - tv0.tv_usec)/1000000 + (double)(tv1.tv_sec - tv0.tv_sec));
  return 0;
}
