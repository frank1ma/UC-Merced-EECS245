
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <time.h>
#include <sys/time.h>


#define RADIUS        3
#define BLOCK_SIZE    256
#define NUM_ELEMENTS  (4096*2)

static void handleError(hipError_t err,const char *file,int line) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),file, line );
        exit( EXIT_FAILURE );
    }
}

#define cudaCheck( err ) (handleError( err, __FILE__, __LINE__ ))

__constant__ int const_arr[NUM_ELEMENTS + 2 * RADIUS];

__global__ void stencil_1d_constant(int *out) {
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    int result = 0;
    for(int offset = -RADIUS; offset <= RADIUS; offset++)
    {
         result += const_arr[gindex + RADIUS  + offset];	       
    }
    out[gindex] = result;
}

int main(){
  unsigned int i;
  int h_in[NUM_ELEMENTS + 2 * RADIUS], h_out[NUM_ELEMENTS];
  int  *d_out;
  
  struct timeval tv1, tv0;

  for( i = 0; i < (NUM_ELEMENTS + 2*RADIUS); ++i ) {
    h_in[i] = 1; // With a value of 1 and RADIUS of 3, all output values should be 7
  }

  cudaCheck( hipMalloc( &d_out, NUM_ELEMENTS * sizeof(int)) );

  hipMemcpyToSymbol(HIP_SYMBOL(const_arr), h_in, (NUM_ELEMENTS + 2*RADIUS) * sizeof(int));

  gettimeofday(&tv0, NULL);

  stencil_1d_constant<<< (NUM_ELEMENTS + BLOCK_SIZE - 1)/BLOCK_SIZE, BLOCK_SIZE >>> (d_out);
   
  gettimeofday(&tv1, NULL);

  cudaCheck(hipPeekAtLastError());
  cudaCheck( hipMemcpy( h_out, d_out, NUM_ELEMENTS * sizeof(int), hipMemcpyDeviceToHost) );
  
  printf("time: %lf\n", double(tv1.tv_usec - tv0.tv_usec)/1000000 + (double)(tv1.tv_sec - tv0.tv_sec));
 
  for( i = 0; i < NUM_ELEMENTS; ++i ){
    if (h_out[i] != 7){
      printf("Element h_out[%d] == %d != 7\n", i, h_out[i]);
      break;
    }
  }
  if (i == NUM_ELEMENTS){
    printf("SUCCESS!\n");
   }
 
  //cudaFree(d_in);
  hipFree(d_out);

  return 0;
}
