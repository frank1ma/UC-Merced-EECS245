#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <time.h>
#include <sys/time.h>

#define RADIUS        3
#define BLOCK_SIZE    256
#define NUM_ELEMENTS  (4096*2)

static void handleError(hipError_t err,const char *file,int line) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),file, line );
        exit( EXIT_FAILURE );
    }
}

#define cudaCheck( err ) (handleError( err, __FILE__, __LINE__ ))

texture<int, 1, hipReadModeElementType> texture_arr;

__global__ void stencil_1d_texture(int* out){
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    int result = 0;
        for (int offset = -RADIUS ; offset <= RADIUS ; offset++) {
            result += tex1Dfetch(texture_arr, gindex + RADIUS + offset);		
        }
	out[gindex] = result;

}

int main(){
  unsigned int i;
  int h_in[NUM_ELEMENTS + 2 * RADIUS], h_out[NUM_ELEMENTS];
  int *d_in, *d_out;
  
  struct timeval tv1, tv0;

  for( i = 0; i < (NUM_ELEMENTS + 2*RADIUS); ++i ){
    h_in[i] = 1; 
  }
  cudaCheck( hipMalloc( &d_in, (NUM_ELEMENTS + 2*RADIUS) * sizeof(int)) );
  cudaCheck( hipMalloc( &d_out, NUM_ELEMENTS * sizeof(int)) );

  cudaCheck( hipMemcpy( d_in, h_in, (NUM_ELEMENTS + 2*RADIUS) * sizeof(int), hipMemcpyHostToDevice) );
  hipBindTexture(0, texture_arr, d_in, (NUM_ELEMENTS + 2*RADIUS) * sizeof(int));  
  
  gettimeofday(&tv0, NULL);

  stencil_1d_texture<<< (NUM_ELEMENTS + BLOCK_SIZE - 1)/BLOCK_SIZE, BLOCK_SIZE >>> (d_out);
  
  gettimeofday(&tv1, NULL);

  hipUnbindTexture(texture_arr);   

  cudaCheck(hipPeekAtLastError());  
  cudaCheck( hipMemcpy( h_out, d_out, NUM_ELEMENTS * sizeof(int), hipMemcpyDeviceToHost) );

  printf("time: %lf\n", double(tv1.tv_usec - tv0.tv_usec)/1000000 + (double)(tv1.tv_sec - tv0.tv_sec));

  for( i = 0; i < NUM_ELEMENTS; ++i ) {
    if (h_out[i] != 7){
      printf("Element h_out[%d] == %d != 7\n", i, h_out[i]);
      break;
    }
  }
  if (i == NUM_ELEMENTS) {
    printf("SUCCESS!\n");
  }
 
  hipFree(d_in);
  hipFree(d_out);

  return 0;
}
